#include "hip/hip_runtime.h"
/*
 * yolo_layer.cu
 *
 * This code was originally written by wang-xinyu under MIT license.
 * I took it from:
 *
 *     https://github.com/wang-xinyu/tensorrtx/tree/master/yolov4
 *
 * and made necessary modifications.
 *
 * - JK Jung
 */

#include "YoloLayerPlugin.h"

#include <cassert>
#include <iostream>
#include <hip/hip_math_constants.h>

namespace
{
// Write values into buffer
template<typename T>
void write(char*& buffer, const T& val)
{
	*reinterpret_cast<T*>(buffer) = val;
	buffer += sizeof(T);
}

// Read values from buffer
template<typename T>
void read(const char*& buffer, T& val)
{
	val = *reinterpret_cast<const T*>(buffer);
	buffer += sizeof(T);
}

#define CHECK_YOLO(status)                                    \
	do                                                        \
	{                                                         \
		auto ret = status;                                    \
		if (ret != 0)                                         \
		{                                                     \
			std::cerr << "Cuda failure in file '" << __FILE__ \
					  << "' line " << __LINE__                \
					  << ": " << ret << std::endl;            \
			abort();                                          \
		}                                                     \
	} while (0)
} // namespace

namespace nvinfer1
{
YoloLayerPlugin::YoloLayerPlugin(int yolo_width, int yolo_height, int num_anchors, float* anchors, int num_classes, int input_width, int input_height, float scale_x_y)
{
	mYoloWidth  = yolo_width;
	mYoloHeight = yolo_height;
	mNumAnchors = num_anchors;
	memcpy(mAnchorsHost, anchors, num_anchors * 2 * sizeof(float));
	mNumClasses  = num_classes;
	mInputWidth  = input_width;
	mInputHeight = input_height;
	mScaleXY     = scale_x_y;

	CHECK_YOLO(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
	CHECK_YOLO(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));
}

YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
{
	const char* d = reinterpret_cast<const char*>(data);
#ifndef NDEBUG
	const char* a = d;
#endif
	read(d, mThreadCount);
	read(d, mYoloWidth);
	read(d, mYoloHeight);
	read(d, mNumAnchors);
	memcpy(mAnchorsHost, d, MAX_ANCHORS * 2 * sizeof(float));
	d += MAX_ANCHORS * 2 * sizeof(float);
	read(d, mNumClasses);
	read(d, mInputWidth);
	read(d, mInputHeight);
	read(d, mScaleXY);

	CHECK_YOLO(hipMalloc(&mAnchors, MAX_ANCHORS * 2 * sizeof(float)));
	CHECK_YOLO(hipMemcpy(mAnchors, mAnchorsHost, mNumAnchors * 2 * sizeof(float), hipMemcpyHostToDevice));

	assert(d == a + length);
}

void YoloLayerPlugin::serialize(void* buffer) const NOEXCEPT
{
	char* d = static_cast<char*>(buffer);
#ifndef NDEBUG
	char* a = d;
#endif
	write(d, mThreadCount);
	write(d, mYoloWidth);
	write(d, mYoloHeight);
	write(d, mNumAnchors);
	memcpy(d, mAnchorsHost, MAX_ANCHORS * 2 * sizeof(float));
	d += MAX_ANCHORS * 2 * sizeof(float);
	write(d, mNumClasses);
	write(d, mInputWidth);
	write(d, mInputHeight);
	write(d, mScaleXY);

	assert(d == a + getSerializationSize());
}

size_t YoloLayerPlugin::getSerializationSize() const NOEXCEPT
{
	return sizeof(mThreadCount) +
		   sizeof(mYoloWidth) + sizeof(mYoloHeight) +
		   sizeof(mNumAnchors) + MAX_ANCHORS * 2 * sizeof(float) +
		   sizeof(mNumClasses) +
		   sizeof(mInputWidth) + sizeof(mInputHeight) +
		   sizeof(mScaleXY);
}

void YoloLayerPlugin::terminate() NOEXCEPT
{
	CHECK_YOLO(hipFree(mAnchors));
}

Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) NOEXCEPT
{
	assert(index == 0);
	assert(nbInputDims == 1);
	assert(inputs[0].d[0] == (mNumClasses + 5) * mNumAnchors);
	assert(inputs[0].d[1] == mYoloHeight);
	assert(inputs[0].d[2] == mYoloWidth);
	// output detection results to the channel dimension
	int totalsize = mYoloWidth * mYoloHeight * mNumAnchors * sizeof(Yolo::Detection) / sizeof(float);
	return Dims3(totalsize, 1, 1);
}

// Clone the plugin
IPluginV2IOExt* YoloLayerPlugin::clone() const NOEXCEPT
{
	YoloLayerPlugin* p = new YoloLayerPlugin(mYoloWidth, mYoloHeight, mNumAnchors, (float*)mAnchorsHost, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
	p->setPluginNamespace(mPluginNamespace);
	return p;
}

inline __device__ float sigmoidGPU(float x)
{
	return 1.0f / (1.0f + __expf(-x));
}

inline __device__ float scale_sigmoidGPU(float x, float scale)
{
	return scale * sigmoidGPU(x) - (scale - 1.0f) * 0.5f;
}

// CalDetection(): This kernel processes 1 yolo layer calculation.  It
// distributes calculations so that 1 GPU thread would be responsible
// for each grid/anchor combination.
// NOTE: The output (x, y, w, h) are between 0.0 and 1.0
//       (relative to orginal image width and height).
__global__ void CalDetection(const float* input, float* output, int yolo_width, int yolo_height, int num_anchors,
							 const float* anchors, int num_classes, int input_w, int input_h, float scale_x_y)
{
	int idx         = threadIdx.x + blockDim.x * blockIdx.x;
	Yolo::Detection* det  = ((Yolo::Detection*)output) + idx;
	int total_grids = yolo_width * yolo_height;
	if (idx >= total_grids * num_anchors) return;

	int anchor_idx         = idx / total_grids;
	idx                    = idx - total_grids * anchor_idx;
	int info_len           = 5 + num_classes;
	const float* cur_input = input + anchor_idx * (info_len * total_grids);

	int class_id;
	float max_cls_logit = -HIP_INF_F; // minus infinity
	for (int i = 5; i < info_len; ++i)
	{
		float l = cur_input[idx + i * total_grids];
		if (l > max_cls_logit)
		{
			max_cls_logit = l;
			class_id      = i - 5;
		}
	}
	float max_cls_prob = sigmoidGPU(max_cls_logit);
	float box_prob     = sigmoidGPU(cur_input[idx + 4 * total_grids]);

	int row = idx / yolo_width;
	int col = idx % yolo_width;

	det->bbox[0] = (col + scale_sigmoidGPU(cur_input[idx + 0 * total_grids], scale_x_y)) / yolo_width;  // [0, 1]
	det->bbox[1] = (row + scale_sigmoidGPU(cur_input[idx + 1 * total_grids], scale_x_y)) / yolo_height; // [0, 1]
	det->bbox[2] = __expf(cur_input[idx + 2 * total_grids]) * anchors[2 * anchor_idx] / input_w;        // [0, 1]
	det->bbox[3] = __expf(cur_input[idx + 3 * total_grids]) * anchors[2 * anchor_idx + 1] / input_h;    // [0, 1]

	det->bbox[0] -= det->bbox[2] / 2; // shift from center to top-left
	det->bbox[1] -= det->bbox[3] / 2;

	det->det_confidence   = box_prob;
	det->class_id         = class_id;
	det->class_confidence = max_cls_prob;
}

void YoloLayerPlugin::forwardGpu(const float* const* inputs, float* output, hipStream_t stream, int batchSize)
{
	int num_elements = batchSize * mNumAnchors * mYoloWidth * mYoloHeight;

	CalDetection<<<(num_elements + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>(inputs[0], output, mYoloWidth, mYoloHeight, mNumAnchors, (const float*)mAnchors, mNumClasses, mInputWidth, mInputHeight, mScaleXY);
}

#if NV_TENSORRT_MAJOR >= 8
int32_t YoloLayerPlugin::enqueue(int32_t batchSize, void const* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) NOEXCEPT
#else   // NV_TENSORRT_MAJOR < 8
int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
#endif  // NV_TENSORRT_MAJOR
{
	forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
	return 0;
}

YoloPluginCreator::YoloPluginCreator()
{
	mPluginAttributes.clear();

	mFC.nbFields = mPluginAttributes.size();
	mFC.fields   = mPluginAttributes.data();
}

const char* YoloPluginCreator::getPluginName() const NOEXCEPT
{
	return "YoloLayer_TRT";
}

const char* YoloPluginCreator::getPluginVersion() const NOEXCEPT
{
	return "1";
}

const PluginFieldCollection* YoloPluginCreator::getFieldNames() NOEXCEPT
{
	return &mFC;
}

IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) NOEXCEPT
{
	assert(!strcmp(name, getPluginName()));
	const PluginField* fields = fc->fields;
	int yolo_width, yolo_height, num_anchors = 0;
	float anchors[MAX_ANCHORS * 2];
	int num_classes;
	int input_width, input_height;
	float scale_x_y = 1.0;

	for (int i = 0; i < fc->nbFields; ++i)
	{
		const char* attrName = fields[i].name;
		if (!strcmp(attrName, "yoloWidth"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			yolo_width = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "yoloHeight"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			yolo_height = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "numAnchors"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			num_anchors = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "numClasses"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			num_classes = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "inputWidth"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			input_width = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "inputHeight"))
		{
			assert(fields[i].type == PluginFieldType::kINT32);
			input_height = *(static_cast<const int*>(fields[i].data));
		}
		else if (!strcmp(attrName, "anchors"))
		{
			assert(num_anchors > 0 && num_anchors <= MAX_ANCHORS);
			assert(fields[i].type == PluginFieldType::kFLOAT32);
			memcpy(anchors, static_cast<const float*>(fields[i].data), num_anchors * 2 * sizeof(float));
		}
		else if (!strcmp(attrName, "scaleXY"))
		{
			assert(fields[i].type == PluginFieldType::kFLOAT32);
			scale_x_y = *(static_cast<const float*>(fields[i].data));
		}
	}
	assert(yolo_width > 0 && yolo_height > 0);
	assert(anchors[0] > 0.0f && anchors[1] > 0.0f);
	assert(num_classes > 0);
	assert(input_width > 0 && input_height > 0);
	assert(scale_x_y >= 1.0);

	YoloLayerPlugin* obj = new YoloLayerPlugin(yolo_width, yolo_height, num_anchors, anchors, num_classes, input_width, input_height, scale_x_y);
	obj->setPluginNamespace(mNamespace.c_str());
	return obj;
}

IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) NOEXCEPT
{
	YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
	obj->setPluginNamespace(mNamespace.c_str());
	return obj;
}

PluginFieldCollection YoloPluginCreator::mFC{};
std::vector<PluginField> YoloPluginCreator::mPluginAttributes;
} // namespace nvinfer1
